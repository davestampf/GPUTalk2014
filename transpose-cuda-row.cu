#include "hip/hip_runtime.h"
/*
 * transpose an array - using a cuda device, 1 thread/row
 */

#include <stdio.h>
#include <stdlib.h>

__global__ void cudaTransposeRow(float* out, float *in, int size);
void startClock(char*);
void stopClock(char*);
void printClock(char*);

#define DIM 1024 

int main(int argc, char** argv) {

	float *h_in;
	float *h_out;

	h_in = (float*) malloc(DIM*DIM*sizeof(float));
	h_out =(float*) malloc(DIM*DIM*sizeof(float));

	void *d_in;
	void *d_out;

	hipMalloc(&d_in,DIM*DIM*sizeof(float));
	hipMalloc(&d_out,DIM*DIM*sizeof(float));

	int value = 1;
	for (int i = 0; i < DIM; i++) {
		for (int j = 0; j < DIM; j++) {
			h_in[i + j*DIM] = value++;
		}
	}

	startClock("copy in");
	hipMemcpy(d_in,h_in,DIM*DIM*sizeof(float),hipMemcpyHostToDevice);
	stopClock("copy in");

	startClock("compute");
	cudaTransposeRow<<<1,1024>>>((float*)d_out,(float*)d_in,DIM);
	hipDeviceSynchronize();
	stopClock("compute");
		
	startClock("copy out");
	hipMemcpy(h_out,d_out,DIM*DIM*sizeof(float),hipMemcpyDeviceToHost);
	stopClock("copy out");

	// sanity check

	for (int i = 0; i < DIM; i++) {
		for (int j = 0; j < DIM; j++) {
			if (h_in[i + j*DIM] != h_out[i*DIM + j]) {
				printf("ERROR");
				exit(1);
			}
		}
	}
	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);

	printClock("copy in");
	printClock("compute");
	printClock("copy out");
}

__global__ void cudaTransposeRow(float* out, float* in, int size) {
	int row = threadIdx.x;
	
	for (int j = 0; j < size; j++) {
		out[j + row*size] = in[row + j*size];
	}
}	

