/*
 * Map the square function sqeezing all of this data through 1 CPU.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void nonCudaMap(float* out, float *in, int size);
void startClock(char*);
void stopClock(char*);
void printClock(char*);

float square(float x) {
	return x*x;
}

int main(int argc, char** argv) {

	if (argc < 2) {
		printf("Usage: %s #-of-floats\n",argv[0]);
		exit(1);
	}
	int size = atoi(argv[1]);
	printf("size = %d\n",size);

	float *h_in;
	float *h_out;

	h_in = (float*) malloc(size*sizeof(float));
	h_out =(float*) malloc(size*sizeof(float));

	for (int i = 0; i < size; i++) {
		h_in[i] = i;
	}

	startClock("compute");
	nonCudaMap(h_out,h_in,size);
	stopClock("compute");
		
	for (int i = 0; i < size; i++) {
		printf("%f -> %f\n",h_in[i],h_out[i]);
	}

	free(h_in);
	free(h_out);

	printClock("compute");
}

void nonCudaMap(float* out, float* in, int size) {
	for (int i = 0; i < size; i++) {
		out[i] = square(in[i]);
	}
}	

