
#include <hip/hip_runtime.h>
/*
 * squaring map kernel that runs in 1 block
 */

/*
 * runs on and callable from the device
 */

__device__ float square(float x) {
	return x*x;
}

/*
 * runs on device, callable from anywhere
 */

__global__ void map(float* out, float* in, int size) {
	int index = threadIdx.x;
	if (index >= size) return;
	out[index] = square(in[index]);
}

