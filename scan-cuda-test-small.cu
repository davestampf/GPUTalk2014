#include "hip/hip_runtime.h"
/*
 * a simple test of the scan kernel.
 */

#include <stdio.h>
#include <stdlib.h>

__global__ void cudaScan(float* out, float *in, int size);
void startClock(char*);
void stopClock(char*);
void printClock(char*);

int main(int argc, char** argv) {

	if (argc < 2) {
		printf("Usage: %s size-of-array\n",argv[0]);
		exit(1);
	}
	int size = atoi(argv[1]);
	hipDeviceProp_t props;
	hipGetDeviceProperties (&props,0);
	if (size > props.maxThreadsPerBlock) {
		fprintf(stderr,"At most %d elements for small version\n",
			props.maxThreadsPerBlock);
		exit(1);
	}
	printf("size = %d\n",size);

	/* host based data */

	float *h_in;
	float *h_out;

	/* device based data */

	float *d_in;
	float *d_out;

	h_in = (float*) malloc(size*sizeof(float));
	h_out =(float*) malloc(size*sizeof(float));

	hipMalloc(&d_in,size*sizeof(float));
	hipMalloc(&d_out,size*sizeof(float));

	for (int i = 0; i < size; i++) {
		h_in[i] = 1.0;
	}

	startClock("copy data to device");
	hipMemcpy(d_in,h_in,size*sizeof(float),hipMemcpyHostToDevice);
	stopClock("copy data to device");

	startClock("compute");

	cudaScan<<<1,size,2*size*sizeof(float)>>>(d_out,d_in,size);
	hipDeviceSynchronize();

	stopClock("compute");

	startClock("copy data from device");
	hipMemcpy(h_out,d_out,size*sizeof(float),hipMemcpyDeviceToHost);
	stopClock("copy data from device");

	float sum = 0.0f;
	for (int i = 0; i < size; i++) {
		sum += h_in[i];
		printf("%d %f -> %f (%f)\n",i,h_in[i],h_out[i],sum);
	}

	free(h_in);
	free(h_out);

	hipFree(d_in);
	hipFree(d_out);

	printClock("copy data to device");
	printClock("compute");
	printClock("copy data from device");
}

