/*
 * a simple serial scan.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void scan(float* out, float* in, int size);
void startClock(char*);
void stopClock(char*);
void printClock(char*);

int main(int argc, char** argv) {

	if (argc < 2) {
		printf("Usage: %s size-of-array\n",argv[0]);
		exit(1);
	}
	int size = atoi(argv[1]);
	printf("size = %d\n",size);

	/* host based data */

	float *h_in;
	float *h_out;

	h_in = (float*) malloc(size*sizeof(float));
	h_out =(float*) malloc(size*sizeof(float));

	for (int i = 0; i < size; i++) {
		h_in[i] = 1.0;
	}

	startClock("compute");
	scan(h_out,h_in,size);
	stopClock("compute");

	for (int i = 0; i < size; i++) {
		printf("%d %f -> %f\n",i,h_in[i],h_out[i]);
	}

	free(h_in);
	free(h_out);

	printClock("compute");
}

void scan(float* out, float* in, int size) {
	out[0] = 0;
	for (int i = 1; i < size; i++) {
		out[i] = out[i-1] + in[i-1];
	}
}

