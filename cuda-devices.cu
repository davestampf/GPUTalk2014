/*
 * Just how many cuda enabled devices on this machine?
 * Also, what are their properties?
 * 
 * Note - EVERY cuda call returns an error value. While
 * this is vital in real code, it gets in the way of 
 * tutorial code.  I'm showing it here for cudaGetDeviceCount
 * but will omit it for the rest of the tutorial.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(int argc, char** argv) {

	int numberOfDevices;
	hipError_t err;

	err = hipGetDeviceCount(&numberOfDevices);
	if (err != hipSuccess) {
		fprintf(stderr,"fail - hipGetDeviceCount %d\n",err);
		exit(1);
	}
	printf("Number of cuda devices = %d\n",numberOfDevices);

	/* the cudaDeviceProp struct is fairly large - read about it in the
	   docs. */
	for (int dev = 0; dev < numberOfDevices; dev++) {
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props,dev);
		printf("Device # %d\n",dev);
		printf(" name = %s\n",props.name);
		printf(" version = %d.%d\n",props.major,props.minor);
		printf(" total global memory = %ld\n",props.totalGlobalMem);
		printf(" shared Memory/Block = %ld\n",props.sharedMemPerBlock);
		printf(" registers/block = %d\n",props.regsPerBlock);
		printf(" warp size = %d\n",props.warpSize);
		printf(" Max threads/block = %d\n",props.maxThreadsPerBlock);
		printf(" Max Threads Dim = %d x %d x %d\n",props.maxThreadsDim[0],
			props.maxThreadsDim[1],props.maxThreadsDim[2]);
		printf(" Max Grid Size = %d x %d x %d\n",props.maxGridSize[0],
			props.maxGridSize[1],props.maxGridSize[2]);
		printf(" Multi-processor count = %d\n",props.multiProcessorCount);
		printf(" Max Threads/multiprocessor = %d\n",props.maxThreadsPerMultiProcessor);

	}
	return 0;
}
