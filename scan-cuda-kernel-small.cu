
#include <hip/hip_runtime.h>
/*
 * a simple scan program. compute the partial sums of
 * the elements of the input array. This version uses
 * only one block
 */

__global__ void cudaScan(float* d_out, float* d_in, int n) {

	// shared array allocated by the launch of the kernel

	extern __shared__ float temp[];
	
	int threadId = threadIdx.x;
	if (threadId >=  n) return;

	int fromBuffer = 1;
	int toBuffer = 0;

	// make a local copy of the data

	temp[threadId] = d_in[threadId];
	__syncthreads();


	int maxOffset =(int)ceil(log2(1.0f*n));
	maxOffset = pow(2.0f,1.0f*maxOffset);
	for (int offset = 1; offset < maxOffset; offset *= 2) {

		fromBuffer = 1-fromBuffer;
		toBuffer = 1-toBuffer;
		if (threadId >= offset) {
			temp[toBuffer*n + threadId] = temp[fromBuffer*n + threadId - offset] +
				temp[fromBuffer*n + threadId]; 
		} else {
			temp[toBuffer*n+ threadId] = temp[fromBuffer*n + threadId];
		}
		__syncthreads();
	}

	d_out[threadId] = temp[toBuffer*n + threadId];
}

