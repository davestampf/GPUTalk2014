/*
 * compute 0 + 1 + 2 + ... + 1023 using cuda - in a bad way
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void sum(int *result) {
	//*result = *result + threadIdx.x;
	atomicAdd(result,threadIdx.x);
}

int main() {

	int h_result = 0;
	void *d_result;

	hipMalloc(&d_result,sizeof(int));
	hipMemcpy(d_result,&h_result,sizeof(int),hipMemcpyHostToDevice);
	
	sum<<<1,1024>>>((int*) d_result);

	hipMemcpy(&h_result,d_result,sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_result);

	printf("We computed %d - should have been %d\n",h_result,1024*1023/2);
}

