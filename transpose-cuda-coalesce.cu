#include "hip/hip_runtime.h"
/*
 * transpose an array - using a cuda device, use coalescing
 */

#include <stdio.h>
#include <stdlib.h>

__global__ void cudaTransposeCoalesce(float* out, float *in, int size);
void startClock(char*);
void stopClock(char*);
void printClock(char*);

#define DIM 1024 

int main(int argc, char** argv) {

	float *h_in;
	float *h_out;

	h_in = (float*) malloc(DIM*DIM*sizeof(float));
	h_out =(float*) malloc(DIM*DIM*sizeof(float));

	void *d_in;
	void *d_out;

	hipMalloc(&d_in,DIM*DIM*sizeof(float));
	hipMalloc(&d_out,DIM*DIM*sizeof(float));

	int value = 1;
	for (int i = 0; i < DIM; i++) {
		for (int j = 0; j < DIM; j++) {
			h_in[i + j*DIM] = value++;
		}
	}

	startClock("copy in");
	hipMemcpy(d_in,h_in,DIM*DIM*sizeof(float),hipMemcpyHostToDevice);
	stopClock("copy in");

	startClock("compute");

	int tileSize = 8;
	int tempMem = tileSize*tileSize*sizeof(float);
	dim3 blocks(128,128,1);
	dim3 threads(8,8);

	cudaTransposeCoalesce<<<blocks,threads,tempMem>>>((float*)d_out,(float*)d_in,DIM);
	hipDeviceSynchronize();
	stopClock("compute");
		
	startClock("copy out");
	hipMemcpy(h_out,d_out,DIM*DIM*sizeof(float),hipMemcpyDeviceToHost);
	stopClock("copy out");

	// sanity check

	for (int i = 0; i < DIM; i++) {
		for (int j = 0; j < DIM; j++) {
			if (h_in[i + j*DIM] != h_out[i*DIM + j]) {
				printf("ERROR");
				exit(1);
			}
		}
	}
	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);

	printClock("copy in");
	printClock("compute");
	printClock("copy out");
}

__global__ void cudaTransposeCoalesce(float* out, float* in, int size) {

	__shared__ float shared[1024];


	// starting points inside the input data

	int iStart = blockDim.x*blockIdx.x;
	int jStart = blockDim.y*blockIdx.y;

	// let adjacent threads pick up adjacent items from input
	// transpose on the fly

	float data = in[iStart + threadIdx.x + (jStart + threadIdx.y)*size];
	shared[threadIdx.y + threadIdx.x*blockDim.x] = data;

	__syncthreads();

	// ok now put them back to out, but travel with the grain!

	int temp = jStart;
	jStart = iStart;
	iStart = temp;

	out[iStart + threadIdx.x + (jStart + threadIdx.y)*size] = shared[threadIdx.x + threadIdx.y*blockDim.y];
}

