
#include <hip/hip_runtime.h>
/*
 * squaring map kernel that runs in N linear blocks
 */

/*
 * runs on and callable from the device
 */

__device__ float square(float x) {
	return x*x;
}

__global__ void map(float* out, float* in, int size) {
	int index = blockDim.x*blockIdx.x + threadIdx.x;
	if (index >= size) return;
	out[index] = square(in[index]);
}

