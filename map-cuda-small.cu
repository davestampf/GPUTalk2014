#include "hip/hip_runtime.h"
/*
 * A very simple cuda implementation of map
 */

#include <stdio.h>
#include <stdlib.h>

__global__ void map(float* out, float* in, int size); 

void startClock(char*);
void stopClock(char*);
void printClock(char*);

int main(int argc, char** argv) {

        if (argc < 2) {
                printf("Usage: %s #-of-floats\n",argv[0]);
                exit(1);
        }
        int size = atoi(argv[1]);
        printf("size = %d\n",size);
	
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props,0);
	if (size > props.maxThreadsPerBlock) {
		fprintf(stderr,"Max size for the small model is %d\n",
			props.maxThreadsPerBlock);
		exit(1);
	}

	void *d_in;	// device data
	void *d_out;
	float *h_in;	// host data
	float *h_out;

	hipMalloc(&d_in,size*sizeof(float));
	hipMalloc(&d_out,size*sizeof(float));
	h_in = (float*) malloc(size*sizeof(float));
	h_out =(float*) malloc(size*sizeof(float));

	for (int i = 0; i < size; i++) {
		h_in[i] = i;
	}

	startClock("copy data to device");	
	hipMemcpy(d_in,h_in,size*sizeof(float),hipMemcpyHostToDevice);
	stopClock("copy data to device");	

	startClock("compute");
	
	// use one block and size threads

	map<<<1,size>>>((float*) d_out,(float*) d_in,size);
	hipDeviceSynchronize();	// forces wait for map to complete

	stopClock("compute");
	
	startClock("copy data to host");
	hipMemcpy(h_out,d_out,size*sizeof(float),hipMemcpyDeviceToHost);
	stopClock("copy data to host");

	for (int i = 0; i < size; i++) {
		printf("%f -> %f\n",h_in[i],h_out[i]);
	}

	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);

	printClock("copy data to device");
	printClock("compute");
	printClock("copy data to host");
}

