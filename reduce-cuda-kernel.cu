
#include <hip/hip_runtime.h>
/*
 * reduction kernel. Initially, each thread will copy 1 item of data
 * from global to shared memory. Then will will do the binary tree dance. 
 */

__global__ void reduce(float* out, float* in, int size) {
	__shared__ float temp[1024];

	int index = blockDim.x*blockIdx.x + threadIdx.x;
	int myId = threadIdx.x; // a value between 0 and 1023

	if (index >= size) return;

	// move data to shared memory for speed

	temp[myId] = in[index];
	__syncthreads();

	int stride = blockDim.x/2;
	while (stride >= 1) {
		if (myId < stride) {
			temp[myId] += temp[myId + stride];
		}
		__syncthreads();
		stride = stride/2;
	}	
	out[blockIdx.x] = temp[0];
}

