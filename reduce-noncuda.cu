/*
 * reduce an array of 1's by the sum
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void nonCudaReduce(float* out, float *in, int size);
void startClock(char*);
void stopClock(char*);
void printClock(char*);

int main(int argc, char** argv) {

	if (argc < 2) {
		printf("Usage: %s #-of-floats\n",argv[0]);
		exit(1);
	}
	int size = atoi(argv[1]);
	printf("size = %d\n",size);

	float *h_in;
	float h_out;

	h_in = (float*) malloc(size*sizeof(float));

	for (int i = 0; i < size; i++) {
		h_in[i] = 1;
	}

	startClock("compute");
	nonCudaReduce(&h_out,h_in,size);
	stopClock("compute");
	
	printf("The sum is %f\n",h_out);	

	free(h_in);

	printClock("compute");
}

void nonCudaReduce(float* out, float* in, int size) {
	*out = 0.0;
	for (int i = 0; i < size; i++) {
		*out += in[i];
	}
}	

