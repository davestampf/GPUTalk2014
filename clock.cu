
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

struct clock {
	char* name;
	long totalTime;
	long startTime;
	struct clock* next;
};

struct clock *clocks = NULL;
long time();

void startClock(char* name) {
	struct clock *cp = clocks;

	while (cp != NULL) {
		if (strcmp(cp->name,name) == 0) {
			clocks->startTime = time();
			return;
		}
		cp = cp->next;
	}
	// if you are here, no match
	cp = (struct clock*)malloc(sizeof(struct clock));
	cp->name = (char*) malloc(strlen(name)+1);
	strcpy(cp->name,name);
	cp->totalTime = 0;
	cp->startTime = time();
	cp->next = clocks;
	clocks = cp;
	return;
}

void stopClock(char* name) {
	struct clock *cp = clocks;
	while (cp && strcmp(cp->name,name)) {
		cp = cp->next;
	}
	if (cp && cp->startTime) {
		cp->totalTime += (time() - cp->startTime);
		cp->startTime = 0;
	}
}

void dump() {
	struct clock *cp = clocks;

	while (cp) {
		printf("%-20s %ld micros\n",cp->name, cp->totalTime);
		cp = cp->next;
	}
}

void printClock(char* name) {
	struct clock *cp = clocks;
	while (cp && strcmp(cp->name,name)) {
		cp = cp->next;
	}
	if (cp) {
		printf("%-20s %ld micros\n",cp->name,cp->totalTime);
	}
}

long time() {
	struct timeval tv;

	gettimeofday(&tv,NULL);
	return 1000000*(tv.tv_sec % (60*60*24*365)) + tv.tv_usec;
}

