/*
 * transpose an array
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void nonCudaTranspose(float* out, float *in, int size);
void startClock(char*);
void stopClock(char*);
void printClock(char*);

#define DIM 1024 

int main(int argc, char** argv) {

	float *h_in;
	float *h_out;

	h_in = (float*) malloc(DIM*DIM*sizeof(float));
	h_out =(float*) malloc(DIM*DIM*sizeof(float));

	int value = 1;
	for (int i = 0; i < DIM; i++) {
		for (int j = 0; j < DIM; j++) {
			h_in[i + j*DIM] = value++;
		}
	}

/*	for (int i = 0; i < DIM; i++) {
		for (int j = 0; j < DIM; j++) {
			printf("%10.2f ",h_in[i+j*DIM]);
		}
		printf("\n");
	}
*/
	startClock("compute");
	nonCudaTranspose(h_out,h_in,DIM);
	stopClock("compute");
		
/*	for (int i = 0; i < DIM; i++) {
		for (int j = 0; j < DIM; j++) {
			printf("%10.2f ",h_out[i+j*DIM]);
		}
		printf("\n");
	}
*/
	free(h_in);
	free(h_out);

	printClock("compute");
}

void nonCudaTranspose(float* out, float* in, int size) {
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			out[j + i*size] = in[i + j*size];
		}
	}
}	

