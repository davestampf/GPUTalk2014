#include "hip/hip_runtime.h"
/*
 * A very simple cuda implementation of reduce. Uses an array of 1024x1024
 * items which are summed into a 1024 array and then summed into a value.
 */

#include <stdio.h>
#include <stdlib.h>

/*
 * this kernel will sum all of the data from in into out - at
 * least as far as the block will carry you
 */

__global__ void reduce(float* out, float* in, int size); 

void startClock(char*);
void stopClock(char*);
void printClock(char*);

int main(int argc, char** argv) {

        int size = 1024*1024;
        printf("size = %d\n",size);
	
	void *d_in;	// device data
	void *d_mid;	// device data - middle results
	void *d_out;	// device data - the answer

	float *h_in;	// host data
	float h_out;

	int numBlocks = 1024;

	hipMalloc(&d_in,size*sizeof(float));
	hipMalloc(&d_mid,numBlocks*sizeof(float));
	hipMalloc(&d_out,sizeof(float));

	h_in = (float*) malloc(size*sizeof(float));

	for (int i = 0; i < size; i++) {
		h_in[i] = 1;
	}

	startClock("copy data to device");	
	hipMemcpy(d_in,h_in,size*sizeof(float),hipMemcpyHostToDevice);
	stopClock("copy data to device");	

	startClock("compute");
	
	// use max threads/block and the required # of blocks AND
	// ask for some shared memory

	reduce<<<1024,1024,1024>>>((float*) d_mid,(float*) d_in,size);
	reduce<<<1,1024,1024>>>((float*)d_out,(float*)d_mid,1024);
	hipDeviceSynchronize();

	stopClock("compute");
	
	startClock("copy data to host");
	h_out = -17;
	hipMemcpy(&h_out,d_out,sizeof(float),hipMemcpyDeviceToHost);
	stopClock("copy data to host");

	printf("The total is %f\n",h_out);
	free(h_in);
	hipFree(d_in);
	hipFree(d_out);

	printClock("copy data to device");
	printClock("compute");
	printClock("copy data to host");
}

