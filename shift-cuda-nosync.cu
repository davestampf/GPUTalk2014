
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void shift(int *xdata, int length) {
	__shared__ int data[1024];

	if (threadIdx.x >= length-1) return;
	data[threadIdx.x] = threadIdx.x;


	if (threadIdx.x > 0) {
		data[threadIdx.x-1] = data[threadIdx.x];
	}

	// copy to global so host can see it

	for (int i = 0; i < length; i++) {
		xdata[i] = data[i];
	}
}

int main() {
	int h_data[1024];
	for (int i = 0; i < 1024; i++) {
		h_data[i] = i;
	}

	void *d_data;

	hipMalloc(&d_data,1024*sizeof(int));
	hipMemcpy(d_data,h_data,1024*sizeof(int),hipMemcpyHostToDevice);

	shift<<<1,1024>>>((int*) d_data,1024);

	hipMemcpy(h_data,d_data,1024*sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_data);

	// lets make sure answer is correct

	for (int i = 0; i < 1023; i++) {
		if (h_data[i] != (i+1)) {
			printf("Differ at position %d value computed %d value expected %d\n",i,h_data[i],i+1);
		}
	}
}
